#include "hip/hip_runtime.h"
//Johnathan Teav
#include <assert.h>

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
void dataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

/* This function embeds PTX code of CUDA to extract bit field from x. 
   "start" is the starting bit position relative to the LSB. 
   "nbits" is the bit field length.
   It returns the extracted bit field as an unsigned integer.
*/
__device__ uint bfe(uint x, uint start, uint nbits)
{
    uint bits;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
    return bits;
}

//Feel free to change the names of the kernels or define more kernels below if necessary

//define the histogram kernel here
__global__ void histogram()
{

}

//define the prefix scan kernel here
//implement it yourself or borrow the code from CUDA samples
__global__ void prefixScan()
{

}

//define the reorder kernel here
__global__ void Reorder()
{

}

int main(int argc, char const *argv[])
{
    int rSize = atoi(argv[1]);
    
    int* r_h; //input array

    hipHostMalloc((void**)&r_h, sizeof(int)*rSize); //use pinned memory in host so it copies to GPU faster
    
    dataGenerator(r_h, rSize, 0, 1);
    
    /* your code */

    hipHostFree(r_h);

    return 0;
}